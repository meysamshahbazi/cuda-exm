#include "hip/hip_runtime.h"


#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>


__global__ void hello_cuda(int i)
{
    i++;
    printf("HEL HEL %d\n",i);
}

int main()
{
    int nx,ny;
    nx = 16;
    ny = 4;
    dim3 block(8,2);
    dim3 grid(nx/block.x, ny/block.y);

    // 
    int i =0;

    hello_cuda<< <grid,block >> > (1);


    hipDeviceSynchronize();
    hipDeviceReset();
     
    return 0;
}
 